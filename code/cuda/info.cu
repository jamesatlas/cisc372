#include <stdio.h> 

int main() {
  int nDevices;

  hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Compute capability: %d.%d\n", prop.major, prop.minor);
    printf("  Number of SMPs: %d\n", prop.multiProcessorCount);
    printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("  Registers per block: %d\n", prop.regsPerBlock);
    printf("  Warp size: %d\n", prop.warpSize);
    printf("  Total global memory: %ld\n", prop.totalGlobalMem);
    printf("  Total constant memory: %ld\n", prop.totalConstMem);
    printf("  Shared memory per block: %ld\n", prop.sharedMemPerBlock);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  }
}
